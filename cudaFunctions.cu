#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <math.h>
#include "myProto.h"

  __global__  void buildHist(int *h, int *temp, int numElements,  int hist_per_thread) {
    int index = blockIdx.x * blockDim.x + threadIdx.x; // current thread index
    int offset_temp = hist_per_thread * RANGE * index; // jumps of 2560

    for (int num = 0; num < RANGE; num++) {
      for (int hist_offset = 0; hist_offset < hist_per_thread; hist_offset++) {
        h[num] += temp[offset_temp + (hist_offset * RANGE) + num];
      } 
    }
  }

  __global__  void buildTemp(int *A, int *temp, int numElements, int hist_per_thread) {
    int index = blockIdx.x * blockDim.x + threadIdx.x; // current thread index
    int offset_A = (numElements / (NUM_BLOCKS * NUM_THREADS)) * index; // jumps of 2500
    int offset_temp = hist_per_thread * RANGE * index; // jumps of 2560

    for (int i = 0; i < numElements / (NUM_BLOCKS * NUM_THREADS); i++) {
      temp[offset_temp + A[offset_A + i]]++;
    }
  }

  __global__  void initHist(int * h) {
    int index = threadIdx.x;
    h[index] = 0;
  }

    __global__  void initTemp(int * temp, int numElements, int hist_per_thread) {
    int index = blockIdx.x * blockDim.x + threadIdx.x; // current thread index
    int offset = hist_per_thread * RANGE * index; // jumps of 2560

    for (int i = 0; i < RANGE * hist_per_thread; i++)
      temp[offset + i] = 0;
  }

int computeOnGPU(int *data, int numElements, int* hist) {

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    int hist_per_thread = (int) ceil((numElements / (NUM_BLOCKS * NUM_THREADS)) / RANGE) + 1;
    // Allocate data on device
    int* d_A = NULL;
    err = hipMalloc((void **)&d_A, numElements * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d cuda (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate hist on device
    int* d_H = NULL;
    err = hipMalloc((void **)&d_H, RANGE * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d cuda (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate temp on device
    int* d_temp = NULL;
    err = hipMalloc((void **)&d_temp, (numElements / (NUM_BLOCKS * NUM_THREADS)) * RANGE * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d cuda (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //Copy data to the device
    err = hipMemcpy(d_A, data, numElements * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
      fprintf(stderr, "Error in line %d cuda (error code %s)!\n", __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    // Initialize hist on device
    initHist<<<1, RANGE>>>(d_H); // 1 block with 256 threads
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d cuda (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Initialize temp on device
    initTemp<<< NUM_BLOCKS, NUM_THREADS >>>(d_temp, numElements, hist_per_thread);
    err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Error in line %d cuda (error code %s)!\n", __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    // Build partial histograms for each thread
    buildTemp<<< NUM_BLOCKS, NUM_THREADS >>>(d_A, d_temp, numElements, hist_per_thread);
    err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Error in line %d cuda (error code %s)!\n", __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    hipDeviceSynchronize();

    // Unify results
    buildHist<<< NUM_BLOCKS, NUM_THREADS >>>(d_H, d_temp, numElements, hist_per_thread);
    err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Error in line %d cuda (error code %s)!\n", __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    hipDeviceSynchronize();

    // Copy the final histogram to the host
    err = hipMemcpy(hist, d_H, RANGE * sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
      fprintf(stderr, "Error in line %d cuda (error code %s)!\n", __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    // Free device global memory
    err = hipFree(d_A);
    if (err != hipSuccess) {
      fprintf(stderr, "Error in line %d cuda (error code %s)!\n", __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

        // Free device global memory
    err = hipFree(d_H);
    if (err != hipSuccess) {
      fprintf(stderr, "Error in line %d cuda (error code %s)!\n", __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    return 0;
}

