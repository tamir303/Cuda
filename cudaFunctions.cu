#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <math.h>
#include "myProto.h"

    /**
     * Unify all partial histograms
     * 
     * @param h The unified histogram
     * @param temp The partial histograms
     * @param numElements Number of given elements in the data array
     * @param hist_per_thread Number of histograms per thread
     * 
     * @return Unifed histogram array
    */
  __global__  void buildHist(int *h, int *temp, int numElements,  int hist_per_thread) {
    int index = blockIdx.x * blockDim.x + threadIdx.x; // current thread index
    int offset_temp = hist_per_thread * RANGE * index; // jumps of 2560

    // For each number and thread, go over all theard's histograms and find current num results
    for (int num = 0; num < RANGE; num++) {
      for (int hist_offset = 0; hist_offset < hist_per_thread; hist_offset++) {
        atomicAdd(&h[num], temp[offset_temp + (hist_offset * RANGE) + num]);
      } 
    }
  }

      /**
     * Build all partial histograms
     * 
     * @param A The data array
     * @param temp The partial histograms
     * @param numElements Number of given elements in the data array
     * @param hist_per_thread Number of histograms per thread
     * 
     * @return Partial histogram array
    */
  __global__  void buildTemp(int *A, int *temp, int numElements, int hist_per_thread) {
    int index = blockIdx.x * blockDim.x + threadIdx.x; // current thread index
    int offset_A = (numElements / (NUM_BLOCKS * NUM_THREADS)) * index; // jumps of 2500
    int offset_temp = hist_per_thread * RANGE * index; // jumps of 2560

    // Calculate each inner thread's histogram on chunk of 2500 numbers
    for (int i = 0; i < numElements / (NUM_BLOCKS * NUM_THREADS); i++) {
      temp[offset_temp + A[offset_A + i]]++;
    }
  }

    /**
     * @param h Initiated unified histogram  array
     * 
     * @return historam array initiated with zeros
    */
  __global__  void initHist(int * h) {
    int index = threadIdx.x;
    h[index] = 0;
  }

  /**
   * Initiate all partial histograms
   * 
   * @param temp Initiated partial histograms
   * @param numElements Number of given elements in the data array
   * @param hist_per_thread Number of histograms per thread
   * 
   * @return Partial histogram initiated with zeros
  */
  __global__  void initTemp(int * temp, int numElements, int hist_per_thread) {
  int index = blockIdx.x * blockDim.x + threadIdx.x; // current thread index
  int offset = hist_per_thread * RANGE * index; // jumps of 2560

  for (int i = 0; i < RANGE * hist_per_thread; i++)
    temp[offset + i] = 0;
  }

int computeOnGPU(int *data, int numElements, int* hist) {

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    int hist_per_thread = (int) ceil((numElements / (NUM_BLOCKS * NUM_THREADS)) / RANGE) + 1;
    // Allocate data on device
    int* d_A = NULL;
    err = hipMalloc((void **)&d_A, numElements * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d cuda (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate hist on device
    int* d_H = NULL;
    err = hipMalloc((void **)&d_H, RANGE * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d cuda (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate temp on device
    int* d_temp = NULL;
    err = hipMalloc((void **)&d_temp, (numElements / (NUM_BLOCKS * NUM_THREADS)) * RANGE * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d cuda (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //Copy data to the device
    err = hipMemcpy(d_A, data, numElements * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
      fprintf(stderr, "Error in line %d cuda (error code %s)!\n", __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    // Initialize hist on device
    initHist<<<1, RANGE>>>(d_H); // 1 block with 256 threads
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d cuda (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Initialize temp on device
    initTemp<<< NUM_BLOCKS, NUM_THREADS >>>(d_temp, numElements, hist_per_thread);
    err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Error in line %d cuda (error code %s)!\n", __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    // Build partial histograms for each thread
    buildTemp<<< NUM_BLOCKS, NUM_THREADS >>>(d_A, d_temp, numElements, hist_per_thread);
    err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Error in line %d cuda (error code %s)!\n", __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    hipDeviceSynchronize();

    // Unify results
    buildHist<<< NUM_BLOCKS, NUM_THREADS >>>(d_H, d_temp, numElements, hist_per_thread);
    err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Error in line %d cuda (error code %s)!\n", __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    hipDeviceSynchronize();

    // Copy the final histogram to the host
    err = hipMemcpy(hist, d_H, RANGE * sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
      fprintf(stderr, "Error in line %d cuda (error code %s)!\n", __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    // Free device global memory
    err = hipFree(d_A);
    if (err != hipSuccess) {
      fprintf(stderr, "Error in line %d cuda (error code %s)!\n", __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

        // Free device global memory
    err = hipFree(d_H);
    if (err != hipSuccess) {
      fprintf(stderr, "Error in line %d cuda (error code %s)!\n", __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    return 0;
}

