#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "myProto.h"

  __global__  void buildHist(int *h, int *array, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    while (tid < size)
      atomicAdd(&h[array[tid]], 1);
      tid += stride;
  }

  __global__  void initHist(int * h) {
    int index = threadIdx.x;
    h[index] = 0;
  }

int computeOnGPU(int *data, int numElements, int* hist) {

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Allocate data on device
    int* d_A = NULL;
    err = hipMalloc((void **)&d_A, numElements * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d cuda (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate hist on device
    int* d_H = NULL;
    err = hipMalloc((void **)&d_H, RANGE * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d cuda (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //Copy data to the device
    err = hipMemcpy(d_A, data, numElements * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
      fprintf(stderr, "Error in line %d cuda (error code %s)!\n", __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    // Initialize hist on device
    initHist<<<1, RANGE>>>(d_H); // 1 block with 256 threads
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d cuda (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Unify the results
    buildHist<<< NUM_BLOCKS, NUM_THREADS >>>(d_H, d_A, numElements);
    err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Error in line %d cuda (error code %s)!\n", __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    // Copy the final histogram to the host
    err = hipMemcpy(hist, d_H, RANGE * sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
      fprintf(stderr, "Error in line %d cuda (error code %s)!\n", __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    // Free device global memory
    err = hipFree(d_A);
    if (err != hipSuccess) {
      fprintf(stderr, "Error in line %d cuda (error code %s)!\n", __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

        // Free device global memory
    err = hipFree(d_H);
    if (err != hipSuccess) {
      fprintf(stderr, "Error in line %d cuda (error code %s)!\n", __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    return 0;
}

